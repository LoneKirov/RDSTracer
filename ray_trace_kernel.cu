#include "hip/hip_runtime.h"
#include <stdio.h>
#include <cmath>
#include "cuda_stuff.h"

__device__ float dot(vec3 v1, vec3 v2) {
    return (v1.x * v2.x) + (v1.y * v2.y) + (v1.z * v2.z);
}

//Sphere Intersection
__device__ float intersect(cuda_sphere_t s, cuda_ray_t r) {
    //Intersection Code
    vec3 l = vec3(s.c.x - r.o.x, s.c.y - r.o.y, s.c.z - r.o.z);
    float s = dot(l, r.d);
    float ll = dot(l, l);
    float rr = c.rr;
    if (s < 0.f && ll > rr) return -1; //sphere is behind us and we're not inside
    float mm = ll-(s*s); //dist from sphere center projected onto ray to sphere center
    if (mm > rr) return -1; //ray misses (sphere center projected onto ray - sphere center > radius)
    float q = sqrtf(rr-mm);
    float t = 0.f;
    if (ll > rr) t = s-q; //we're outside the sphere so return first point
    else t = s+q;
    
    return t;
}

__global__ void RayTraceKernel(cuda_sphere_t spheres[], int spheresSize, cuda_ray_t rays[], cuda_intersection_t intrs[], int width, int height) {
    int row = threadIdx.y + blockIdx.y * blockDim.y;
    int col = threadIdx.x + blockIdx.x * blockDim.x;

    if (row >= height || col >= width) {
        return;
    }

    int idx = (row * width) + col;

    cuda_intersection_t inter();

    for (int i = 0; i < spheresSize; i++) {
        float newT = intersect(spheres[i], rays[idx]);
        if (newT >= 0 && newT < inter.t) {
            inter.t = newT;
            inter.objIndx = i;
        } 
    }

    intrs[idx] = inter;
}
