#include "hip/hip_runtime.h"
#include <stdio.h>
#include <cmath>
#include <cfloat>
#include <thrust/device_vector.h>
#include <thrust/copy.h>
#include "cuda_ray_tracer.h"

using namespace RDST;

__device__ float dot(vec3 &v1, vec3 &v2) {
    return (v1.x * v2.x) + (v1.y * v2.y) + (v1.z * v2.z);
}

__device__ vec3 cross(vec3 &v1, vec3 &v2) {
    vec3 r;
    r.x = v1.y * v2.z - v1.z * v2.y;
    r.y = v1.z * v2.x - v1.x * v2.z;
    r.z = v1.x * v2.y - v1.y * v2.x;
    return r;
}

//Sphere Intersection
__device__ float intersect(cuda_sphere_t &sp, cuda_ray_t &r) {
    //Intersection Code
    vec3 l;
    l.x = sp.c.x - r.o.x;
    l.y = sp.c.y - r.o.y;
    l.z = sp.c.z - r.o.z;
    float s = dot(l, r.d);
    float ll = dot(l, l);
    float rr = sp.rr;
    if (s < 0.f && ll > rr) return -1; //sphere is behind us and we're not inside
    float mm = ll-(s*s); //dist from sphere center projected onto ray to sphere center
    if (mm > rr) return -1; //ray misses (sphere center projected onto ray - sphere center > radius)
    float q = sqrtf(rr-mm);
    float t = 0.f;
    if (ll > rr) t = s-q; //we're outside the sphere so return first point
    else t = s+q;

    return t;
}

__global__ void SphereIntersectKernel(cuda_sphere_t spheres[], int spheresSize,
        cuda_ray_t rays[], cuda_intersection_t intrs[], int rayCount) {
    extern __shared__ cuda_sphere_t sShared[];

    if ((blockIdx.x * blockDim.x + blockIdx.y * blockDim.y * gridDim.x * blockDim.x) > rayCount)
        return;

    int rayPos = threadIdx.y * gridDim.x * blockDim.x + threadIdx.x + blockIdx.x * blockDim.x + blockIdx.y * blockDim.y * gridDim.x * blockDim.x;

    cuda_intersection_t inter;
    inter.objIndx = -1;
    inter.t = FLT_MAX;

    int shPos = threadIdx.y * blockDim.x + threadIdx.x;

    int count = spheresSize / (blockDim.x * blockDim.y);
    int i = 0;
    for (; i < count; i++) {
        sShared[shPos] = spheres[blockDim.x * blockDim.y * i + shPos];
        __syncthreads();
        for (int k = 0; k < blockDim.x * blockDim.y; k++) {
            float newT = intersect(sShared[k], rays[rayPos]);
            if (newT >= 0 && newT < inter.t) {
                inter.t = newT;
                inter.objIndx = i * blockDim.x * blockDim.y + k;
                inter.type = _SPHERE;
            }
        }
        __syncthreads();
    }

    int spill = spheresSize % (blockDim.x * blockDim.y);
    if (shPos < spill)
        sShared[shPos] = spheres[blockDim.x * blockDim.y * i + shPos];
    __syncthreads();
    for (int k = 0; k < spill; k++) {
        float newT = intersect(sShared[k], rays[rayPos]);
        if (newT >= 0 && newT < inter.t) {
            inter.t = newT;
            inter.objIndx = i * blockDim.x * blockDim.y + k;
        }
    }

    if (rayPos < rayCount)
        intrs[rayPos] = inter;
}

//Triangle Intersection
#define VEC_DIFF(vr, v1, v2)    \
{                               \
    vr.x = v1.x - v2.x;         \
    vr.y = v1.y - v2.y;         \
    vr.z = v1.z - v2.z;         \
}
__device__ float intersect(cuda_triangle_t &tr, cuda_ray_t &r) {
    vec3 e1;
    VEC_DIFF(e1, tr.v1, tr.v0);
    vec3 e2;
    VEC_DIFF(e2, tr.v2, tr.v0);
    vec3 p = cross(r.d, e2);
    float a = dot(e1, p);
    if (a == 0.f) return -1;
    float f = 1.f/a;
    vec3 s;
    VEC_DIFF(s, r.o, tr.v0);
    float u = f * dot(s, p);
    if (u < 0.f || u > 1.f) return -1;
    vec3 q = cross(s, e1);
    float v = f * dot(r.d, q);
    if (v < 0.f || u+v > 1.f) return -1;
    return f * dot(e2, q);
}
#undef VEC_DIFF

__global__ void TriangleIntersectKernel(cuda_triangle_t triangles[], int triangleSize,
        cuda_ray_t rays[], cuda_intersection_t intrs[], int rayCount) {
    extern __shared__ cuda_triangle_t tShared[];

    if ((blockIdx.x * blockDim.x + blockIdx.y * blockDim.y * gridDim.x * blockDim.x) > rayCount)
        return;

    int rayPos = threadIdx.y * gridDim.x * blockDim.x + threadIdx.x + blockIdx.x * blockDim.x + blockIdx.y * blockDim.y * gridDim.x * blockDim.x;

    cuda_intersection_t inter;
    inter.objIndx = -1;
    inter.t = FLT_MAX;

    int shPos = threadIdx.y * blockDim.x + threadIdx.x;

    int count = triangleSize / (blockDim.x * blockDim.y);
    int i = 0;
    for (; i < count; i++) {
        tShared[shPos] = triangles[blockDim.x * blockDim.y * i + shPos];
        __syncthreads();
        for (int k = 0; k < blockDim.x * blockDim.y; k++) {
            float newT = intersect(tShared[k], rays[rayPos]);
            if (newT >= 0 && newT < inter.t) {
                inter.t = newT;
                inter.objIndx = i * blockDim.x * blockDim.y + k;
                inter.type = _TRIANGLE;
            }
        }
        __syncthreads();
    }

    int spill = triangleSize % (blockDim.x * blockDim.y);
    if (shPos < spill)
        tShared[shPos] = triangles[blockDim.x * blockDim.y * i + shPos];
    __syncthreads();
    for (int k = 0; k < spill; k++) {
        float newT = intersect(tShared[k], rays[rayPos]);
        if (newT >= 0 && newT < inter.t) {
            inter.t = newT;
            inter.objIndx = i * blockDim.x * blockDim.y + k;
        }
    }

    if (rayPos < rayCount)
        intrs[rayPos] = inter;
}

__host__ intersection_vec RDST::cuda_intersect(const sphere_vec &spheres,
        const triangle_vec &triangles, const ray_vec &rays, int width, int height) {
    static dim3 dimBlock(16, 16);
    static dim3 dimGrid(60, 45);
    static int batchSize = dimBlock.x * dimBlock.y * dimGrid.x * dimGrid.y;
    int count = (width * height) / batchSize;
    int spill = (width * height) % batchSize;
    intersection_vec iVec(rays.size());

    thrust::device_vector<cuda_sphere_t> dSpheres(spheres.begin(), spheres.end());
    cuda_sphere_t *sPtr = thrust::raw_pointer_cast(&(*dSpheres.begin()));

    thrust::device_vector<cuda_triangle_t> dTriangles(triangles.begin(), triangles.end());
    cuda_triangle_t *tPtr = thrust::raw_pointer_cast(&(*dTriangles.begin()));

    ray_vec::const_iterator rVecI = rays.begin();
    intersection_vec::iterator iVecI = iVec.begin();
    for (int i = 0 ; i < count; i++) {
        thrust::device_vector<cuda_ray_t> dRays(rVecI, rVecI + batchSize);

        thrust::device_vector<cuda_intersection_t> dIntersects(batchSize);
        cuda_ray_t *rPtr = thrust::raw_pointer_cast(&(*dRays.begin()));
        cuda_intersection_t *iPtr = thrust::raw_pointer_cast(&(*dIntersects.begin()));

        SphereIntersectKernel<<<dimGrid, dimBlock, sizeof(cuda_sphere_t) * dimBlock.x * dimBlock.y>>>(sPtr,
                spheres.size(), rPtr, iPtr, batchSize);
        TriangleIntersectKernel<<<dimGrid, dimBlock, sizeof(cuda_triangle_t) * dimBlock.x * dimBlock.y>>>(tPtr,
                triangles.size(), rPtr, iPtr, batchSize);

        thrust::copy(dIntersects.begin(), dIntersects.end(), iVecI);
        rVecI += batchSize;
        iVecI += batchSize;
    }

    if (spill) {
        thrust::device_vector<cuda_ray_t> dRays(rVecI, rVecI + spill);

        thrust::device_vector<cuda_intersection_t> dIntersects(spill);
        cuda_ray_t *rPtr = thrust::raw_pointer_cast(&(*dRays.begin()));
        cuda_intersection_t *iPtr = thrust::raw_pointer_cast(&(*dIntersects.begin()));

        SphereIntersectKernel<<<dimGrid, dimBlock, sizeof(cuda_sphere_t) * dimBlock.x * dimBlock.y>>>(sPtr,
                spheres.size(), rPtr, iPtr, spill);
        TriangleIntersectKernel<<<dimGrid, dimBlock, sizeof(cuda_triangle_t) * dimBlock.x * dimBlock.y>>>(tPtr,
                triangles.size(), rPtr, iPtr, spill);

        thrust::copy(dIntersects.begin(), dIntersects.end(), iVecI);
    }

    return iVec;
}
