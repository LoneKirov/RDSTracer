#include "hip/hip_runtime.h"
#include <stdio.h>
#include <cmath>
#include <cfloat>
#include <thrust/device_vector.h>
#include <thrust/copy.h>
#include "cuda_ray_tracer.h"

using namespace RDST;

__device__ float dot(vec3 &v1, vec3 &v2) {
    return (v1.x * v2.x) + (v1.y * v2.y) + (v1.z * v2.z);
}

//Sphere Intersection
__device__ float intersect(cuda_sphere_t &sp, cuda_ray_t &r) {
    //Intersection Code
    vec3 l;
    l.x = sp.c.x - r.o.x;
    l.y = sp.c.y - r.o.y;
    l.z = sp.c.z - r.o.z;
    float s = dot(l, r.d);
    float ll = dot(l, l);
    float rr = sp.rr;
    if (s < 0.f && ll > rr) return -1; //sphere is behind us and we're not inside
    float mm = ll-(s*s); //dist from sphere center projected onto ray to sphere center
    if (mm > rr) return -1; //ray misses (sphere center projected onto ray - sphere center > radius)
    float q = sqrtf(rr-mm);
    float t = 0.f;
    if (ll > rr) t = s-q; //we're outside the sphere so return first point
    else t = s+q;

    return t;
}

__global__ void SphereIntersectKernel(cuda_sphere_t spheres[], int spheresSize,
        cuda_ray_t rays[], cuda_intersection_t intrs[], int rayCount) {
    extern __shared__ cuda_sphere_t shared[];

    if ((blockIdx.x * blockDim.x + blockIdx.y * blockDim.y * gridDim.x * blockDim.x) > rayCount)
        return;

    int rayPos = threadIdx.y * gridDim.x * blockDim.x + threadIdx.x + blockIdx.x * blockDim.x + blockIdx.y * blockDim.y * gridDim.x * blockDim.x;

    cuda_intersection_t inter;
    inter.objIndx = -1;
    inter.t = FLT_MAX;

    int shPos = threadIdx.y * blockDim.x + threadIdx.x;

    int count = spheresSize / (blockDim.x * blockDim.y);
    int i = 0;
    for (; i < count; i++) {
        shared[shPos] = spheres[blockDim.x * blockDim.y * i + shPos];
        __syncthreads();
        for (int k = 0; k < blockDim.x * blockDim.y; k++) {
            float newT = intersect(shared[k], rays[rayPos]);
            if (newT >= 0 && newT < inter.t) {
                inter.t = newT;
                inter.objIndx = i * blockDim.x * blockDim.y + k;
            }
        }
        __syncthreads();
    }

    int spill = spheresSize % (blockDim.x * blockDim.y);
    if (shPos < spill)
        shared[shPos] = spheres[blockDim.x * blockDim.y * i + shPos];
    __syncthreads();
    for (int k = 0; k < spill; k++) {
        float newT = intersect(shared[k], rays[rayPos]);
        if (newT >= 0 && newT < inter.t) {
            inter.t = newT;
            inter.objIndx = i * blockDim.x * blockDim.y + k;
        }
    }

    if (rayPos < rayCount)
        intrs[rayPos] = inter;
}

__host__ intersection_vec RDST::cuda_sphere_intersect(const sphere_vec &spheres, const ray_vec &rays, int width, int height) {
    static dim3 dimBlock(16, 16);
    static dim3 dimGrid(60, 45);
    static int batchSize = dimBlock.x * dimBlock.y * dimGrid.x * dimGrid.y;
    int count = (width * height) / batchSize;
    int spill = (width * height) % batchSize;
    intersection_vec iVec(rays.size());

    thrust::device_vector<cuda_sphere_t> dSpheres(spheres.begin(), spheres.end());
    cuda_sphere_t *sPtr = thrust::raw_pointer_cast(&(*dSpheres.begin()));

    ray_vec::const_iterator rVecI = rays.begin();
    intersection_vec::iterator iVecI = iVec.begin();
    for (int i = 0 ; i < count; i++) {
        thrust::device_vector<cuda_ray_t> dRays(rVecI, rVecI + batchSize);

        thrust::device_vector<cuda_intersection_t> dIntersects(batchSize);
        cuda_ray_t *rPtr = thrust::raw_pointer_cast(&(*dRays.begin()));
        cuda_intersection_t *iPtr = thrust::raw_pointer_cast(&(*dIntersects.begin()));

        SphereIntersectKernel<<<dimGrid, dimBlock, sizeof(cuda_sphere_t) * dimBlock.x * dimBlock.y>>>(sPtr,
                spheres.size(), rPtr, iPtr, batchSize);

        thrust::copy(dIntersects.begin(), dIntersects.end(), iVecI);
        rVecI += batchSize;
        iVecI += batchSize;
    }

    if (spill) {
        thrust::device_vector<cuda_ray_t> dRays(rVecI, rVecI + spill);

        thrust::device_vector<cuda_intersection_t> dIntersects(spill);
        cuda_ray_t *rPtr = thrust::raw_pointer_cast(&(*dRays.begin()));
        cuda_intersection_t *iPtr = thrust::raw_pointer_cast(&(*dIntersects.begin()));

        SphereIntersectKernel<<<dimGrid, dimBlock, sizeof(cuda_sphere_t) * dimBlock.x * dimBlock.y>>>(sPtr,
                spheres.size(), rPtr, iPtr, spill);

        thrust::copy(dIntersects.begin(), dIntersects.end(), iVecI);
    }

    return iVec;
}
