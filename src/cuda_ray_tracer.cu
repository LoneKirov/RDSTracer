#include "hip/hip_runtime.h"
#include <stdio.h>
#include <cmath>
#include <cfloat>
#include <thrust/device_vector.h>
#include <thrust/copy.h>
#include "cuda_ray_tracer.h"

using namespace RDST;

__device__ float dot(vec3 v1, vec3 v2) {
    return (v1.x * v2.x) + (v1.y * v2.y) + (v1.z * v2.z);
}

//Sphere Intersection
__device__ float intersect(cuda_sphere_t sp, cuda_ray_t r) {
    //Intersection Code
    vec3 l;
    l.x = sp.c.x - r.o.x;
    l.y = sp.c.y - r.o.y;
    l.z = sp.c.z - r.o.z;
    float s = dot(l, r.d);
    float ll = dot(l, l);
    float rr = sp.rr;
    if (s < 0.f && ll > rr) return -1; //sphere is behind us and we're not inside
    float mm = ll-(s*s); //dist from sphere center projected onto ray to sphere center
    if (mm > rr) return -1; //ray misses (sphere center projected onto ray - sphere center > radius)
    float q = sqrtf(rr-mm);
    float t = 0.f;
    if (ll > rr) t = s-q; //we're outside the sphere so return first point
    else t = s+q;

    return t;
}

__global__ void RayTraceKernel(cuda_sphere_t spheres[], int spheresSize, cuda_ray_t rays[], cuda_intersection_t intrs[], int width, int height) {
    int row = threadIdx.y + blockIdx.y * blockDim.y;
    int col = threadIdx.x + blockIdx.x * blockDim.x;

    if (row >= height || col >= width) {
        return;
    }

    int idx = (row * width) + col;

    cuda_intersection_t inter;
    inter.objIndx = -1;
    inter.t = FLT_MAX;

    for (int i = 0; i < spheresSize; i++) {
        float newT = intersect(spheres[i], rays[idx]);
        if (newT >= 0 && newT < inter.t) {
            inter.t = newT;
            inter.objIndx = i;
        } 
    }

    intrs[idx] = inter;
}

__host__ intersection_vec RDST::cuda_ray_trace(const sphere_vec &spheres, const ray_vec &rays, int width, int height) {
    thrust::device_vector<cuda_sphere_t> dSpheres(spheres.begin(),
            spheres.end());
    thrust::device_vector<cuda_ray_t> dRays(rays.begin(),
            rays.end());
    thrust::device_vector<cuda_intersection_t> dIntersects(rays.size());

    cuda_sphere_t *sPtr = thrust::raw_pointer_cast(&(*dSpheres.begin()));
    cuda_ray_t *rPtr = thrust::raw_pointer_cast(&(*dRays.begin()));
    cuda_intersection_t *iPtr = thrust::raw_pointer_cast(&(*dIntersects.begin()));
    // Run kernel on spheres
    dim3 dimBlock(16, 16);
    dim3 dimGrid(512, 512);
    RayTraceKernel<<<dimGrid, dimBlock>>>(sPtr, spheres.size(), rPtr, iPtr,
            width, height);

    intersection_vec iVec(dIntersects.size());
    thrust::copy(dIntersects.begin(), dIntersects.end(), iVec.begin());
    return iVec;
}
