#include "hip/hip_runtime.h"
#include <stdio.h>
#include <cmath>
#include <cfloat>
#include <thrust/device_vector.h>
#include <thrust/copy.h>
#include "cuda_ray_tracer.h"

using namespace RDST;

__device__ float dot(vec3 &v1, vec3 &v2) {
    return (v1.x * v2.x) + (v1.y * v2.y) + (v1.z * v2.z);
}

//Sphere Intersection
__device__ float intersect(cuda_sphere_t &sp, cuda_ray_t &r) {
    //Intersection Code
    vec3 l;
    l.x = sp.c.x - r.o.x;
    l.y = sp.c.y - r.o.y;
    l.z = sp.c.z - r.o.z;
    float s = dot(l, r.d);
    float ll = dot(l, l);
    float rr = sp.rr;
    if (s < 0.f && ll > rr) return -1; //sphere is behind us and we're not inside
    float mm = ll-(s*s); //dist from sphere center projected onto ray to sphere center
    if (mm > rr) return -1; //ray misses (sphere center projected onto ray - sphere center > radius)
    float q = sqrtf(rr-mm);
    float t = 0.f;
    if (ll > rr) t = s-q; //we're outside the sphere so return first point
    else t = s+q;

    return t;
}

__global__ void RayTraceKernel(cuda_sphere_t spheres[], int spheresSize,
        cuda_ray_t rays[], cuda_intersection_t intrs[], int width, int height) {
    extern __shared__ cuda_sphere_t shared[];

    if ((blockIdx.y * blockDim.y) >= height || (blockIdx.x * blockDim.x) >= width)
        return;

    int row = threadIdx.y + blockIdx.y * blockDim.y;
    int col = threadIdx.x + blockIdx.x * blockDim.x;

    cuda_intersection_t inter;
    inter.objIndx = -1;
    inter.t = FLT_MAX;

    int shPos = threadIdx.y * blockDim.x + threadIdx.x;

    int count = spheresSize / (blockDim.x * blockDim.y);
    int i = 0;
    for (; i < count; i++) {
        shared[shPos] = spheres[blockDim.x * blockDim.y * i + shPos];
        __syncthreads();
        for (int k = 0; k < blockDim.x * blockDim.y; k++) {
            float newT = intersect(shared[k], rays[row * width + col]);
            if (newT >= 0 && newT < inter.t) {
                inter.t = newT;
                inter.objIndx = i * blockDim.x * blockDim.y + k;
            }
        }
        __syncthreads();
    }

    int spill = spheresSize % (blockDim.x * blockDim.y);
    if (shPos < spill)
        shared[shPos] = spheres[blockDim.x * blockDim.y * i + shPos];
    __syncthreads();
    for (int k = 0; k < spill; k++) {
        float newT = intersect(shared[k], rays[row * width + col]);
        if (newT >= 0 && newT < inter.t) {
            inter.t = newT;
            inter.objIndx = i * blockDim.x * blockDim.y + k;
        }
    }

    if (row < height && col < width)
        intrs[row * width + col] = inter;
}

__host__ intersection_vec RDST::cuda_ray_trace(const sphere_vec &spheres, const ray_vec &rays, int width, int height) {
    thrust::device_vector<cuda_sphere_t> dSpheres(spheres.begin(),
            spheres.end());
    thrust::device_vector<cuda_ray_t> dRays(rays.begin(),
            rays.end());
    thrust::device_vector<cuda_intersection_t> dIntersects(rays.size());

    cuda_sphere_t *sPtr = thrust::raw_pointer_cast(&(*dSpheres.begin()));
    cuda_ray_t *rPtr = thrust::raw_pointer_cast(&(*dRays.begin()));
    cuda_intersection_t *iPtr = thrust::raw_pointer_cast(&(*dIntersects.begin()));
    // Run kernel on spheres
    dim3 dimBlock(16, 16);
    dim3 dimGrid(512, 512);
    RayTraceKernel<<<dimGrid, dimBlock, sizeof(cuda_sphere_t) * dimBlock.x * dimBlock.y>>>(sPtr,
            spheres.size(), rPtr, iPtr, width, height);

    intersection_vec iVec(dIntersects.size());
    thrust::copy(dIntersects.begin(), dIntersects.end(), iVec.begin());
    return iVec;
}
