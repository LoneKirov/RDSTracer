#include "hip/hip_runtime.h"

/**
* File: RDSTracer.cpp
* --------------------
* The actual Ray Tracer Implementation!
* Author: Ryan Schmitt
*/

#include "RDSTracer.h"
#include "ProgressBar.h"
#include <iostream>

namespace RDST
{
   void Tracer::RayTrace(const SceneDescription& scene, Image& image)
   {
      //Create rays
      std::vector<RayPtr> rays(GenerateRays(scene.cam(), image));
      
      //Run CUDA
      std::cout << "\nRunning CUDA Intersections...";

      //Create CUDA data structures
      // spheres
      int spheresLen = scene.spheres().size();
      cuda_sphere_t* cudaHostSphereArray = new cuda_sphere_t[spheresLen];
      initCudaSpheres(cudaHostSphereArray, scene.spheres());

      //Allocate and copy spheres to device
      cuda_sphere_t* cudaDeviceSphereArray;
      hipMalloc((void**)&cudaDeviceSphereArray, spheresLen * sizeof(cuda_sphere_t));
      hipMemcpy(cudaDeviceSphereArray, cudaHostSphereArray, spheresLen * sizeof(cuda_sphere_t), 
              hipMemcpyHostToDevice);

      // rays
      int raysLen = rays.size();
      cuda_ray_t* cudaHostRayArray = new cuda_ray_t[raysLen];
      initCudaRays(cudaHostRayArray, rays);

      //Allocate and copy rays to device
      cuda_ray_t* cudaDeviceRayArray;
      hipMalloc((void**)&cudaDeviceRayArray, raysLen * sizeof(cuda_ray_t));
      hipMemcpy(cudaDeviceRayArray, cudaHostRayArray, raysLen * sizeof(cuda_ray_t), 
              hipMemcpyHostToDevice);

      // intersections
      cuda_intersection_t* cudaHostIntersectionArray = new cuda_intersection_t[raysLen](); //1 intersection per ray

      //Allocate and copy rays to device
      cuda_intersection_t* cudaDeviceIntersectionArray;
      hipMalloc((void**)&cudaDeviceIntersectionArray, raysLen * sizeof(cuda_intersection_t));
      hipMemcpy(cudaDeviceIntersectionArray, cudaHostIntersectionArray, raysLen * sizeof(cuda_intersection_t), 
              hipMemcpyHostToDevice);

      dim3 dimBlock(16, 16);
      dim3 dimGrid(512, 512);

      // Run kernel on spheres
      RayTraceKernel<<<dimGrid, dimBlock>>>(cudaDeviceSphereArray, spheresLen, cudaDeviceRayArray, cudaDeviceIntersectionArray, image.getWidth(), image.getHeight());

      hipMemcpy(cudaHostIntersectionArray, cudaDeviceIntersectionArray, raysLen * sizeof(cuda_intersection_t), 
              hipMemcpyDeviceToHost);

      hipFree(cudaDeviceSphereArray);
      hipFree(cudaDeviceRayArray);
      hipFree(cudaDeviceIntersectionArray);

      std::cout << "Done!\n";

      //Trace non-CUDA obects and shade
      std::cout << "Tracing Rays\n";
      for (unsigned int rayi=0; rayi < rays.size(); ++rayi) {
         //Intersect each ray against all objects
         Intersection* pIntrs = RayObjectsIntersect(*rays[rayi], scene.objs());
         //Shade on hit
         if (pIntrs->hit && cudaHostIntersectionArray[rayi].t > pIntrs->t) {
            ShadePixel(image.get(rayi), scene, *pIntrs);
         }
         else if (cudaHostIntersectionArray[rayi].objIndx > -1) {
            float cudaT = cudaHostIntersectionArray[rayi].t;
            glm::vec3 hitPoint = rays[rayi]->o + (rays[rayi]->d*cudaT);
            glm::vec3 center = scene.spheres()[cudaHostIntersectionArray[rayi].objIndx]->getCenter();
            glm::mat3 normalXform = scene.spheres()[cudaHostIntersectionArray[rayi].objIndx]->getNormalXform(); //May not need this as there's no model xforms for this lab.
            glm::vec3 n = normalXform * glm::normalize(hitPoint-center);
            Surface s = Surface(scene.spheres()[cudaHostIntersectionArray[rayi].objIndx]->getColor(), scene.spheres()[cudaHostIntersectionArray[rayi].objIndx]->getFinish());
            Intersection cudaIntrs = Intersection(true, cudaT, hitPoint, n, s);
            ShadePixel(image.get(rayi), scene, cudaIntrs);
         }
         delete pIntrs;
         //Progress Bar: update every 10,000 rays
         if (rayi % 10000 == 0) UpdateProgress(int(float(rayi)/rays.size()*100.f));
      }
      UpdateProgress(100);
      std::cout << "\n";
   }

   /* Assumes pSphereArr has enough space to fit all spheres */
   void Tracer::initCudaSpheres(cuda_sphere_t pSphereArr[], const std::vector<SpherePtr>& spheres)
   {
      std::vector<SpherePtr>::const_iterator cit = spheres.begin();
      for (int i=0; cit != spheres.end(); ++cit, ++i) {
         pSphereArr[i].rr = (*cit)->getRadiusSquared(); //only radius squared is needed, and why not just store it rather than compute it?
         pSphereArr[i].c = vec3((*cit)->getCenter().x, (*cit)->getCenter().y, (*cit)->getCenter().z);
      }
   }

   void Tracer::initCudaRays(cuda_ray_t pRayArr[], const std::vector<RayPtr>& rays)
   {
      std::vector<RayPtr>::const_iterator cit = rays.begin();
      for (int i=0; cit != rays.end(); ++cit, ++i) {
         pRayArr[i].d = vec3((*cit)->d.x, (*cit)->d.y, (*cit)->d.z);
         pRayArr[i].o = vec3((*cit)->o.x, (*cit)->o.y, (*cit)->o.z);
      }
   }

   std::vector<RayPtr> Tracer::GenerateRays(const Camera& cam, const Image& image)
   {
      std::cout << "Generating Rays\n";
      std::vector<RayPtr> rays;
      float h = image.getHeight();
      float w = image.getWidth();
      float r = glm::length(cam.getRight())*0.5f;
      float l = -r;
      float t = glm::length(cam.getUp())*0.5f;
      float b = -t;
      for (int y=0; y<h; y++) {
         for (int x=0; x<w; x++) {
            //Get view coords
            float u = l+((r-l)*(x+0.5f)/w);
            float v = b+((t-b)*(y+0.5f)/h);
            //Create Ray
            glm::vec3 rayOrigin(0.f,0.f,0.f); //view space
            glm::vec3 rayDir(u,v,1.f);
            glm::mat4 matViewWorld(glm::vec4(glm::normalize(cam.getRight()),0.f), glm::vec4(cam.getUp(),0.f), glm::vec4(cam.getDir(),0.f), glm::vec4(cam.getPos(),1.f));
            rayOrigin = glm::vec3(matViewWorld * glm::vec4(rayOrigin, 1.f)); //convert to world space
            rayDir = glm::normalize(glm::vec3(matViewWorld * glm::vec4(rayDir, 0.f)));
            rays.push_back(RayPtr(new Ray(rayDir, rayOrigin)));
            //Progress Bar: update every 10,000 rays
            if ((int)(y*w+x) % 10000 == 0) {
               UpdateProgress(int((y*w+x)/(w*h)*100.f));
            }
         }
      }
      UpdateProgress(100);
      std::cout << "\n";
      return rays;
   }

   Intersection* Tracer::RayObjectsIntersect(Ray& ray, const std::vector<GeomObjectPtr>& objs)
   {
      Intersection* pRetIntrs = new Intersection(); //defaults to hit=false
      //Intersect loop over all objects to find the closest hit
      std::vector<GeomObjectPtr>::const_iterator cit = objs.begin();
      for (; cit != objs.end(); ++cit) {
         Intersection* pIntrs = (*cit)->intersect(ray);
         //Check for closer, valid, hit
         if (pIntrs != NULL && //implies hit
             pIntrs->t < ray.tCur &&
             pIntrs->t < ray.tMax &&
             pIntrs->t > ray.tMin) {
                ray.tCur = pIntrs->t; //set new current t
                delete pRetIntrs;
                pRetIntrs = pIntrs; //it's closer; grab it!
         }
      }
      return pRetIntrs;
   }

   void Tracer::ShadePixel(Pixel& p, const SceneDescription& scene, const Intersection& intrs)
   {
      //Required Vars
      PointLight& light = *scene.lights().at(0); //TODO: add more lights
      //Ambient
      glm::vec3 ambient(intrs.surf.finish.getAmbient() * intrs.surf.color * light.getColor());
      //Diffuse and Specular
      glm::vec3 pointToLight = light.getPos()-intrs.p;
      //Ray shadowRay = Ray(glm::normalize(pointToLight), intrs.p+(0.01f*intrs.n)); //Note to self: needed to move the shadow ray off the origin object a bit
      //shadowRay.tMax = glm::length(pointToLight);
      glm::vec3 diffuse(0.f);
      glm::vec3 specular(0.f);
      //Intersection* pShadowIntrs = RayObjectsIntersect(shadowRay, scene.objs());
      if (true /*!pShadowIntrs->hit*/) {
         //diffuse calcs
         glm::vec3 l = glm::normalize(light.getPos()-intrs.p);
         float diff = glm::max(0.f, glm::dot(intrs.n, l));
         diffuse = glm::vec3(diff * intrs.surf.finish.getDiffuse() * intrs.surf.color * light.getColor());
         //specular calcs
         glm::vec3 v = glm::normalize(scene.cam().getPos()-intrs.p);
         glm::vec3 h = glm::normalize(l+v);
         float spec = glm::max(0.f, glm::dot(intrs.n, h));
         specular = glm::vec3(powf(spec,1.f/intrs.surf.finish.getRoughness()) * intrs.surf.color * intrs.surf.finish.getSpecular() * light.getColor());
      }
      //delete pShadowIntrs;
      //Put it all together and blend
      glm::vec4 src(ambient + diffuse + specular,1.f);
      glm::vec4 dst = p.rgba();
      dst = (src*src.a) + (dst*(1-src.a)); //glBlendFunc(GL_SRC_ALPHA, GL_ONE_MINUS_SRC_ALPHA)
      p.set(dst);
   }
}
